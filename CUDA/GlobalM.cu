#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

__device__ __constant__ int maxIteraci = 500;
__device__ __constant__ float minPosun = 0.001f;
__device__ __constant__ Hodnoty hodnotyD;

/**
 * CUDA kernel function, ktera dela meanShift segmentaci
 * @param *img ukazatel na zpracovavany obrazek
 * @param *res ukazatel na vysldek
 *
 */
__global__ void doMeanShift(const unsigned char *img, unsigned char *res ) {

	int xP = blockIdx.x*blockDim.x + threadIdx.x;
	int yP = blockIdx.y*blockDim.y + threadIdx.y;

	if(xP >= hodnotyD.x || yP >= hodnotyD.y)
		return;
	float windowSum = 0.0f;
	float xyzSum[3];
    for(int i =0; i < 3 ; i++)
    	xyzSum[i] = 0.0f;
    float sum = 0.0f;
    float posun = 0.0f;
    int iterace = 0;
    //xxd = (x - xi)/h etc.
    float xxd = 0.0f, yyd = 0.0f, zzd = 0.0f;
    unsigned char childColor = 0;
    float x = xP;
    float y = yP;
    float rootColor = img[xP*hodnotyD.y + yP];
    int iA, iB, jA, jB;
    float rozdil;


    while(true){

    	//okno
    	iA = ( x - hodnotyD.radius - 1);
    	iB = ( x + hodnotyD.radius + 1);
    	jA = ( y - hodnotyD.radius - 1);
    	jB = ( y + hodnotyD.radius + 1);

        //okraje obrazu
        if( iA < 0 ) iA = 0;
        if( jA < 0 ) jA = 0;
        if( iB >= hodnotyD.x ) iB = hodnotyD.x;
        if( jB >= hodnotyD.y ) jB = hodnotyD.y;

        for( int i = iA ; i < iB ; i++ )
        {
            for( int j = jA ; j < jB ; j ++)
            {
                sum = 0.0f;
                //kruh
                if( (i - x) * (i - x) + (j - y ) * (j - y) <= hodnotyD.radiusE2 )
                {
                    childColor = img[ i * hodnotyD.y + j];
                    rozdil =  rootColor - childColor;
 
                        xxd = (x - i) / (float)hodnotyD.radius;
                        yyd = (y - j) / (float)hodnotyD.radius;
                        zzd = rozdil / (float)hodnotyD.colorDiff;

                        sum = xxd*xxd + yyd*yyd + zzd*zzd;

                        //kernel(Epanechnikov)
                        if( sum >= 1.0f )
                            sum = 0.0f;
                        else
                            sum = 0.75f * ( 1.0f - sum * sum );

                        windowSum += sum;

                        xyzSum[0] += sum*i;
                        xyzSum[1] += sum*j;
                        xyzSum[2] += sum*childColor;
                    

                }//kruh

            }//for
        }//for

        xyzSum[0] /= windowSum;
        xyzSum[1] /= windowSum;
        xyzSum[2] /= windowSum;
        posun = sqrt((xyzSum[0] - x) * (xyzSum[0] - x) + (xyzSum[1] - y) * (xyzSum[1] - y) + (xyzSum[2] - rootColor) * (xyzSum[2] - rootColor));
        iterace++;

        if( iterace >= maxIteraci || posun < minPosun ){
        	res[xP*hodnotyD.y + yP] = xyzSum[2];
        	break;
        }

        x = xyzSum[0];
        y = xyzSum[1];
        rootColor = xyzSum[2];

        xyzSum[0] = xyzSum[1] = xyzSum[2] = windowSum = 0.0f;

     }//wile
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

/**
 * fce kopiruje data na kartu, kopiruje vysledek do pameti, vraci cas provedeni kernelu v milsec.
 */
float mainCu( unsigned char *img, unsigned char *result, Hodnoty *hodnoty ) {

	//mereni casu
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	size_t Size =  hodnoty->x*hodnoty->y;

	//ukazatele na device mem.
	unsigned char * img_d, *result_d;
	//Hodnoty *val_d;
	dim3 threadsPerBlock(16, 16);
	int xBlock = (hodnoty->x / threadsPerBlock.x);
	if( hodnoty->x % threadsPerBlock.x ) xBlock++;
	int yBlock = (hodnoty->y / threadsPerBlock.y);
	if( hodnoty->y % threadsPerBlock.y ) yBlock++;
	dim3 numBlocks( xBlock , yBlock );

	hipEventRecord(start, 0);

	//alokace
	hipMalloc((void **) &img_d, sizeof(unsigned char)*Size);
	hipMalloc((void **) &result_d, sizeof(unsigned char)*Size);
	//hipMalloc((void **) &val_d, sizeof(Hodnoty));

	checkCUDAError("hipMalloc to device");

	//nakopiruju do dev
	hipMemcpy(img_d, img, ( Size*sizeof(unsigned char) ), hipMemcpyHostToDevice);
	hipMemcpy(result_d, result, ( Size*sizeof(unsigned char) ), hipMemcpyHostToDevice);
	//Hodnoty h = *hodnoty;
	//const. mem.
	hipMemcpyToSymbol(HIP_SYMBOL(hodnotyD), hodnoty, ( sizeof(Hodnoty) ), 0,hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy to device");

	doMeanShift<<<numBlocks, threadsPerBlock>>>(img_d, result_d);
	checkCUDAError("doMeanShift(kernel call)");

	// block until the device has completed
	hipDeviceSynchronize();

	checkCUDAError("hipDeviceSynchronize");

	//po skonceni konci mereni casu
	hipEventSynchronize(stop);

	// device to host copy
	hipMemcpy( result, result_d, Size*sizeof(unsigned char), hipMemcpyDeviceToHost );

	checkCUDAError("hipMemcpy to host");
	hipEventRecord(stop, 0);

	//uvolneni pameti
	hipFree(img_d);
	hipFree(result_d);
	//hipFree(val_d);

	//vysledny cas
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);//vypocet casu, presnost okolo 0.5 microseconds
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;

}
